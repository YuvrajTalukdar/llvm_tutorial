#include<iostream>
#include<hip/hip_runtime.h>

using namespace std;

class name
{
    private:
    int x;
    public:
    void hello()
    {

    }
    int y;
};

__global__ void test108()
{
    printf("\nhello world from device");
}

int main()
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,0);
    cout<<"Device Name: "<<deviceProp.name<<", sm_count: "<<deviceProp.multiProcessorCount<<endl;
    
    int cudaVersion;
    hipError_t cudaStatus = hipRuntimeGetVersion(&cudaVersion);
    if(cudaStatus == hipSuccess) 
    {
        cout << "CUDA Version: " << cudaVersion << std::endl;
    } 
    else 
    {
        cerr << "Error getting CUDA version: " << hipGetErrorString(cudaStatus) << std::endl;
    }
    
    int blocks=1,threads=512;
    test108<<<blocks,threads>>>();
    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) 
    {
        cout<<"CUDA error: "<< hipGetErrorString(error);
    }

    return 0;
}
