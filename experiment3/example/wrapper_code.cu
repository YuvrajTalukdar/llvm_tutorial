/**
*   CS6023: GPU Programming 
*   Assignment 1
*   
*   Please don't change any existing code in this file.
*
*   You can add your code whereever needed. Please add necessary memory APIs
*   for your implementation. Use cudaFree() to free up memory as soon as you're
*   done with an allocation. This will ensure that you don't run out of memory 
*   while running large test cases. Use the minimum required memory for your 
*   implementation. DO NOT change the kernel configuration parameters.
*/

#include <chrono>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>

using std::cin;
using std::cout;

struct dim6_index
{
    long int block_x,block_y,block_z;
    long int thread_x,thread_y,thread_z;
};

void f00109()
{

}
__host__ __device__
void foo108(int x)
{
    printf("%d",x);
}

__device__
long int get_id(dim6_index dim_index,dim3 block_dim,dim3 grid_dim,long int required_itteration,long int ittr_index)
{
    long int thread_id=blockDim.y*block_dim.x*required_itteration*dim_index.thread_z+
                        block_dim.x*required_itteration*dim_index.thread_y+
                        required_itteration*dim_index.thread_x+
                        ittr_index;
    long int block_id=grid_dim.x*grid_dim.y*dim_index.block_z+
                        grid_dim.x*dim_index.block_y+
                        dim_index.block_x;
    long int id=(block_dim.x*block_dim.y*block_dim.z*required_itteration)*block_id+thread_id;
    //printf("\nthreadId: %ld block_id: %ld id: %ld",thread_id,block_id,id);
    return id;
}

__device__
long int get_id(dim6_index dim_index,dim3 block_dim,dim3 grid_dim)
{
    long int thread_id=blockDim.y*block_dim.x*dim_index.thread_z+
                        block_dim.x*dim_index.thread_y+
                        dim_index.thread_x;
    long int block_id=grid_dim.x*grid_dim.y*dim_index.block_z+
                        grid_dim.x*dim_index.block_y+
                        dim_index.block_x;
    long int id=(block_dim.x*block_dim.y*block_dim.z)*block_id+thread_id;
    //printf("\nthreadId: %ld block_id: %ld id: %ld",thread_id,block_id,id);
    return id;
}

//divident=divisor*quotient+remainder
__device__
dim6_index get_dim6(long int id,dim3 block_dim,struct dim3 grid_dim)
{
    struct dim6_index dim_index;
    dim_index.block_z = id / (grid_dim.y * grid_dim.x * block_dim.z * block_dim.y * block_dim.x);
    dim_index.block_y = (id % (grid_dim.y * grid_dim.x * block_dim.z * block_dim.y * block_dim.x)) / (grid_dim.x * block_dim.z * block_dim.y * block_dim.x);
    dim_index.block_x = (id % (grid_dim.x * block_dim.z * block_dim.y * block_dim.x)) / (block_dim.z * block_dim.y * block_dim.x);
    dim_index.thread_z = (id % (block_dim.z * block_dim.y * block_dim.x)) / (block_dim.y * block_dim.x);
    dim_index.thread_y = (id % (block_dim.y * block_dim.x)) / block_dim.x;
    dim_index.thread_x = id % block_dim.x;
    return dim_index;
}

__device__
void calc_loop_itteration(dim3 gridDim_original,dim3 blockDim_original,long int &quotient,long int &remainder)
{
    quotient=(blockDim_original.x*blockDim_original.y*blockDim_original.z*gridDim_original.x*gridDim_original.y*gridDim_original.z) / 
            (blockDim.x*blockDim.y*blockDim.z*gridDim.x*gridDim.y*gridDim.z);
    remainder=(blockDim_original.x*blockDim_original.y*blockDim_original.z*gridDim_original.x*gridDim_original.y*gridDim_original.z) %
            (blockDim.x*blockDim.y*blockDim.z*gridDim.x*gridDim.y*gridDim.z);
}

__device__
void calc_loop_itteration(long int new_dim,long int original_dim,long int &required_itteration,long int &remainder)
{
    required_itteration=0;
    if(new_dim<original_dim)
    {   
        required_itteration=original_dim/new_dim;
        remainder=original_dim%new_dim;
    }
    else
    {   required_itteration=1;}   
}

__device__
void CalculateHadamardProduct(long int* A, long int* B, int N,long int blockDim_x,long int threadIdx_x,long int blockIdx_x) 
{
    // TODO: Write your kernel here
    long int index=blockIdx_x*blockDim_x+threadIdx_x;
    long int cols=index%N;
    long int rows=index/N;
    if(rows*N+cols<N*N)
    {   A[rows*N+cols]=A[rows*N+cols]*B[cols*N+rows];}
}

//demonstration of non perfect division
__global__ 
void CalculateHadamardProduct_boilerplate(long int* A, long int* B, int N,dim3 gridDim_original,dim3 blockDim_original)
{
    long int required_ittration,remaining_itteration;
    calc_loop_itteration(gridDim_original,blockDim_original,required_ittration,remaining_itteration);
    
    dim6_index dim_index_current;
    dim_index_current.thread_x=threadIdx.x;    
    dim_index_current.thread_y=threadIdx.y;
    dim_index_current.thread_z=threadIdx.z;
    dim_index_current.block_x=blockIdx.x;
    dim_index_current.block_y=blockIdx.y;
    dim_index_current.block_z=blockIdx.z;   
    long int id_current;
    dim6_index dim_index_original;

    for(long int a=0;a<required_ittration;a++)
    {
        id_current=get_id(dim_index_current,blockDim,gridDim,required_ittration,a);
        dim_index_original=get_dim6(id_current,blockDim_original,gridDim_original);
        
        //printf("\nblock: %ld %ld %ld thread: %ld %ld %ld",dim_index_original.block_x,dim_index_original.block_y,dim_index_original.block_z,dim_index_original.thread_x,dim_index_original.thread_y,dim_index_original.thread_z);

        CalculateHadamardProduct(A,B,N,blockDim_original.x,dim_index_original.thread_x,dim_index_original.block_x);
    }
    short id=get_id(dim_index_current,blockDim,gridDim);
    if(id<remaining_itteration)
    {
        id_current=(gridDim.x*gridDim.y*gridDim.z*blockDim.x*blockDim.y*blockDim.z*required_ittration)+id;
        //printf("\nid_current: %d remaining: %ld",id_current,remaining_itteration);
        dim_index_original=get_dim6(id_current,blockDim_original,gridDim_original);
        CalculateHadamardProduct(A,B,N,blockDim_original.x,dim_index_original.thread_x,dim_index_original.block_x);
    }
}

__device__
void FindWeightMatrix(long int* A, long int* B, int N,long int blockDim_y,long int blockDim_x,long int threadIdx_y,long int threadIdx_x,long int blockIdx_x) 
{
    // TODO: Write your kernel here
    long int thread_id=threadIdx_y*blockDim_y+threadIdx_x;
    long int index=blockIdx_x*(blockDim_x*blockDim_y)+thread_id;
    long int cols=index%N;
    long int rows=index/N;

    if((index)<(N*N))
    {
        if(A[rows*N+cols]<B[rows*N+cols])
        {   A[rows*N+cols]=B[rows*N+cols];}
    }
}

__global__
void FindWeightMatrix_boilerplate(long int* A, long int* B, int N,dim3 gridDim_original,dim3 blockDim_original)
{
    long int required_ittration,remaining_itteration;
    calc_loop_itteration(gridDim_original,blockDim_original,required_ittration,remaining_itteration);

    dim6_index dim_index_current;
    dim_index_current.thread_x=threadIdx.x;    
    dim_index_current.thread_y=threadIdx.y;
    dim_index_current.thread_z=threadIdx.z;
    dim_index_current.block_x=blockIdx.x;
    dim_index_current.block_y=blockIdx.y;
    dim_index_current.block_z=blockIdx.z;   
    long int id_current;
    dim6_index dim_index_original;

    for(long int a=0;a<required_ittration;a++)
    {
        id_current=get_id(dim_index_current,blockDim,gridDim,required_ittration,a);
        dim_index_original=get_dim6(id_current,blockDim_original,gridDim_original);
        
        FindWeightMatrix(A,B,N,blockDim_original.y,blockDim_original.x,dim_index_original.thread_y,dim_index_original.thread_x,dim_index_original.block_x);
    }
    short id=get_id(dim_index_current,blockDim,gridDim);
    if(id<remaining_itteration)
    {
        id_current=(gridDim.x*gridDim.y*gridDim.z*blockDim.x*blockDim.y*blockDim.z*required_ittration)+id;
        dim_index_original=get_dim6(id_current,blockDim_original,gridDim_original);
        
        FindWeightMatrix(A,B,N,blockDim_original.y,blockDim_original.x,dim_index_original.thread_y,dim_index_original.thread_x,dim_index_original.block_x);
    }
}

__device__
void CalculateFinalMatrix(long int* A, long int* B, int N,long int gridDim_y,long int blockDim_y,long int blockDim_x,long int threadIdx_y,long int threadIdx_x,long int blockIdx_y,long int blockIdx_x)
{
    // TODO: Write your kernel here
    long int thread_id=threadIdx_y*blockDim_y+threadIdx_x;
    long int block_id=blockIdx_y*gridDim_y+blockIdx_x;
    long int index=block_id*(blockDim_x*blockDim_y)+thread_id;
    long int cols_d=index%(N*2);
    long int rows_d=index/(N*2);
    if((index)<(2*N*2*N))
    {
        long int rows=rows_d%(N);
        long int cols=cols_d%(N);
        B[rows_d*2*N+cols_d]=B[rows_d*2*N+cols_d]*A[rows*N+cols];
    }
}

__global__
void CalculateFinalMatrix_boilerplate(long int* A, long int* B, int N,dim3 gridDim_original,dim3 blockDim_original)
{
    long int required_ittration,remaining_itteration;
    calc_loop_itteration(gridDim_original,blockDim_original,required_ittration,remaining_itteration);

    dim6_index dim_index_current;
    dim_index_current.thread_x=threadIdx.x;    
    dim_index_current.thread_y=threadIdx.y;
    dim_index_current.thread_z=threadIdx.z;
    dim_index_current.block_x=blockIdx.x;
    dim_index_current.block_y=blockIdx.y;
    dim_index_current.block_z=blockIdx.z;   
    long int id_current;
    dim6_index dim_index_original;

    for(long int a=0;a<required_ittration;a++)
    {
        id_current=get_id(dim_index_current,blockDim,gridDim,required_ittration,a);
        dim_index_original=get_dim6(id_current,blockDim_original,gridDim_original);
        
        CalculateFinalMatrix(A,B,N,gridDim_original.y,blockDim_original.y,blockDim_original.x,dim_index_original.thread_y,dim_index_original.thread_x,dim_index_original.block_y,dim_index_original.block_x);
    }
    short id=get_id(dim_index_current,blockDim,gridDim);
    if(id<remaining_itteration)
    {
        id_current=(gridDim.x*gridDim.y*gridDim.z*blockDim.x*blockDim.y*blockDim.z*required_ittration)+id;
        dim_index_original=get_dim6(id_current,blockDim_original,gridDim_original);
        
        CalculateFinalMatrix(A,B,N,gridDim_original.y,blockDim_original.y,blockDim_original.x,dim_index_original.thread_y,dim_index_original.thread_x,dim_index_original.block_y,dim_index_original.block_x);
    }
}

void print_mat(long int *mat,int size)
{
    for(int a=0;a<size;a++)
    {
        for(int b=0;b<size;b++)
        {
            printf("%d,",mat[a*size+b]);
        }
        printf("\n");
    }
}

int main(int argc, char** argv) {


    int N;
    cin >> N;
    long int* A = new long int[N * N];
    long int* B = new long int[N * N];
    long int* C = new long int[N * N];
    long int* D = new long int[2 * N * 2 * N];


    for (long int i = 0; i < N * N; i++) {
        cin >> A[i];
    }

    for (long int i = 0; i < N * N; i++) {
        cin >> B[i];
    }

    for (long int i = 0; i < N * N; i++) {
        cin >> C[i];
    }

    for (long int i = 0; i < 2 * N * 2 * N; i++) {
        cin >> D[i];
    }

    /**
     * 
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     * 
    */
    hipSetDevice(0);
    long int* d_A;
    long int* d_B;
    long int* d_C;
    long int* d_D;

    //printf("\nA:\n");
    //print_mat(A,N);
    dim3 threadsPerBlock(1024, 1, 1);
    dim3 blocksPerGrid(ceil(N * N / 1024.0), 1, 1);
    
    hipMalloc(&d_A,sizeof(long int)*N*N);
    hipMalloc(&d_B,sizeof(long int)*N*N);
    hipMemcpy(d_A,A,sizeof(long int)*N*N,hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,sizeof(long int)*N*N,hipMemcpyHostToDevice);
    
    auto start = std::chrono::high_resolution_clock::now();
    //CalculateHadamardProduct<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, N);//original
    dim3 threadsPerBlock_new(3,3,4);
    dim3 blocksPerGrid_new(2,1,12);
    printf("\nkernel 1");
    printf("\ntotal_threads_new: %d",blocksPerGrid_new.x*blocksPerGrid_new.y*blocksPerGrid_new.z*threadsPerBlock_new.x*blocksPerGrid_new.y*blocksPerGrid_new.z);
    printf("\ntotal_threads_old: %d",blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z*threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z);
    CalculateHadamardProduct_boilerplate<<<blocksPerGrid_new, threadsPerBlock_new>>>(d_A, d_B, N,blocksPerGrid,threadsPerBlock);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed1 = end - start;
    
    hipMemcpy(A,d_A,sizeof(long int)*N*N,hipMemcpyDeviceToHost);
    hipFree(d_B);
    //cudaMemcpy(B,d_B,sizeof(long int)*N*N,cudaMemcpyDeviceToHost);
    //printf("\nHarmad A:\n");
    //print_mat(A,N);
    //printf("\nC:\n");
    //print_mat(C,N);
    
    
    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(N * N / 1024.0), 1, 1);
    
    hipMalloc(&d_C,sizeof(long int)*N*N);
    hipMemcpy(d_C,C,sizeof(long int)*N*N,hipMemcpyHostToDevice);
    
    start = std::chrono::high_resolution_clock::now();
    //FindWeightMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, N);//original
    threadsPerBlock_new=dim3(3,13,3);
    blocksPerGrid_new=dim3(17,6,2);
    printf("\nkernel 2");
    printf("\ntotal_threads_new: %d",blocksPerGrid_new.x*blocksPerGrid_new.y*blocksPerGrid_new.z*threadsPerBlock_new.x*blocksPerGrid_new.y*blocksPerGrid_new.z);
    printf("\ntotal_threads_old: %d",blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z*threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z);
    FindWeightMatrix_boilerplate<<<blocksPerGrid_new, threadsPerBlock_new>>>(d_A, d_C, N,blocksPerGrid, threadsPerBlock);//original
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed2 = end - start;
    
    hipMemcpy(A,d_A,sizeof(long int)*N*N,hipMemcpyDeviceToHost);
    hipFree(d_C);
    //printf("\nweight:\n");
    //print_mat(A,N);
    //printf("\nD:\n");
    //print_mat(D,N*2);
    

    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(2 * N / 32.0), ceil(2 * N / 32.0), 1);

    hipMalloc(&d_D,sizeof(long int)*2*N*2*N);
    hipMemcpy(d_D,D,sizeof(long int)*N*2*N*2,hipMemcpyHostToDevice);
    
    start = std::chrono::high_resolution_clock::now();
    //CalculateFinalMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_D, N);//original
    
    threadsPerBlock_new=dim3(17,1,2);
    blocksPerGrid_new=dim3(1,3,5);
    printf("\nkernel 3");
    printf("\ntotal_threads_new: %d",blocksPerGrid_new.x*blocksPerGrid_new.y*blocksPerGrid_new.z*threadsPerBlock_new.x*blocksPerGrid_new.y*blocksPerGrid_new.z);
    printf("\ntotal_threads_old: %d",blocksPerGrid.x*blocksPerGrid.y*blocksPerGrid.z*threadsPerBlock.x*threadsPerBlock.y*threadsPerBlock.z);
    CalculateFinalMatrix_boilerplate<<<blocksPerGrid_new, threadsPerBlock_new>>>(d_A, d_D, N, blocksPerGrid, threadsPerBlock);//original
    
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed3 = end - start;

    hipMemcpy(D,d_D,sizeof(long int)*N*2*N*2,hipMemcpyDeviceToHost);
    hipFree(d_A);
    // Make sure your final output from the device is stored in d_D.

    /**
     * 
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     * 
    */

    hipMemcpy(D, d_D, 2 * N * 2 * N * sizeof(long int), hipMemcpyDeviceToHost);

    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < 2 * N; i++) {
            for (long int j = 0; j < 2 * N; j++) {
                file << D[i * 2 * N + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2 << elapsed2.count() << "\n";
        file2 << elapsed3.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}
